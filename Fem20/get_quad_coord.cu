#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "headers/hemi.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers/Common.h"

// assert() is only supported // for devices of compute capability 2.0 and higher 
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200) 
#undef  assert 
#define assert(arg) 
#endif

__constant__ double c_tau;
__constant__ double c_tau_to_current_time_level;
__constant__ double c_lb;
__constant__ double c_rb;
__constant__ double c_ub;
__constant__ double c_bb;
__constant__ double c_tau_b;
__constant__ double c_pi_half;
__constant__ int c_x_length;
__constant__ int c_n;


// ядро написано для того, чтобы избежать копирования кусочка x и y на девайс во время вычисления координат
// Посчитаем объем глобальной памяти
// Пусть N  - число элементов внутренней матрицы
//   Тогда для запуска этого ядра необходимо
//   2*N - для хранения X, Y
//   2*N - для хранения координат точки alpha на предыдущем временном слое
//   2*N - для хранения координат точки betta на предыдущем временном слое
//   2*N - для хранения координат точки gamma на предыдущем временном слое
//   2*N - для хранения координат точки theta на предыдущем временном слое
//   итого = 10*N глобальной памяти
// Дополнительно храня 8*N элементов, мы сможем избавиться от копирования x и y на карту 
// и сэкономить 2*N памяти в основном ядре расчетов
// Но самое главное - это сократить нагрузку на регистры в основном ядре, избежав спиллинга регистров в локальную память
// Это позволит запускать на расчет бОльшие сетки. Глобальную память легче маштабировать нежели регистры
__global__ void get_square_coord(double* first1, double* second1, double* third1,
	double* first2, double* second2, double* third2)
{
	for (int opt = hemiGetElementOffset(); opt < c_n; opt += hemiGetElementStride())
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;
		double x, y, h = 1. / (c_x_length + 1);
		
		// A
		x = (h*(i - 1) + h*i) / 2.;
		y = (h*(j - 1) + h*j) / 2.;
		first1[2 * opt] = first2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		first1[2 * opt + 1] = first2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// B
		x = (h*(i + 1) + h*i) / 2.;
	    //	y = (h*(j - 1) + h*j) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second1[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second1[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// C
		//x = (a_x[i + 1] + a_x[i]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		y = (h*(j + 1) + h*j) / 2.;
		third1[2 * opt] = third2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		third1[2 * opt + 1] = third2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// D
		x = (h*(i - 1) + h*i) / 2.;
		//y = (a_y[j + 1] + a_y[j]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
	}
}

void convert(TriangleResult* result, double* first1, double* second1, double* third1,
	double* first2, double* second2, double* third2, int n)
{
	int k = 0;

	for (int i = 0; i < n; i++)
	{
		result->f[i].first[0] = first1[k];
		result->f[i].first[1] = first1[k + 1];
		result->f[i].second[0] = second1[k];
		result->f[i].second[1] = second1[k + 1];
		result->f[i].third[0] = third1[k];
		result->f[i].third[1] = third1[k + 1];
		result->s[i].first[0] = first2[k];
		result->s[i].first[1] = first2[k + 1];
		result->s[i].second[0] = second2[k];
		result->s[i].second[1] = second2[k + 1];
		result->s[i].third[0] = third2[k];
		result->s[i].third[1] = third2[k + 1];
		k += 2;
	}

}

float get_quad_coord(TriangleResult* result, ComputeParameters* p, int gridSize, int blockSize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	size_t size(0), n(0);
	double temp(0);
	n = p->get_inner_matrix_size();


	float elapsedTime;
	double *first1 = NULL, *second1 = NULL, *third1 = NULL, *first2 = NULL, *second2 = NULL, *third2 = NULL;

	// Start record
	hipEventRecord(start, 0);

	hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &p->tau, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_lb), &p->lb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_rb), &p->rb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_bb), &p->bb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_ub), &p->ub, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_n), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_x_length), &result->x_length, sizeof(int));

	temp = (1. + p->currentTimeLevel * p->tau) / 10.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_to_current_time_level), &temp, sizeof(double));

	temp = p->b * p->tau;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_b), &temp, sizeof(double));

	temp = C_pi_device / 2.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pi_half), &temp, sizeof(double));
	
	size = 2 * sizeof(double)*n;
	checkCuda(hipMallocManaged(&first1, size));
	checkCuda(hipMallocManaged(&second1, size));
	checkCuda(hipMallocManaged(&third1, size));
	checkCuda(hipMallocManaged(&first2, size));
	checkCuda(hipMallocManaged(&second2, size));
	checkCuda(hipMallocManaged(&third2, size));

	get_square_coord<< <gridSize, blockSize >> >(first1, second1, third1, first2, second2, third2);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	convert(result, first1, second1, third1, first2, second2, third2, n);

	hipFree(first1);
	hipFree(second1);
	hipFree(third1);
	hipFree(first2);
	hipFree(second2);
	hipFree(third2);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipDeviceReset();
	return elapsedTime;
}