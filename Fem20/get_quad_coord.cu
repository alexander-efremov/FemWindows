#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "headers/hemi.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers/Common.h"

// assert() is only supported // for devices of compute capability 2.0 and higher 
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200) 
#undef  assert 
#define assert(arg) 
#endif

__constant__ double c_tau;
__constant__ double c_h;
__constant__ double c_tau_to_current_time_level;
__constant__ double c_lb;
__constant__ double c_rb;
__constant__ double c_ub;
__constant__ double c_bb;
__constant__ double c_tau_b;
__constant__ double c_pi_half;
__constant__ int c_x_length;
__constant__ int c_n;

__global__ void get_square_coord(double* first1, double* second1, double* third1,
	double* first2, double* second2, double* third2)
{
	for (int opt = hemiGetElementOffset(); opt < c_n; opt += hemiGetElementStride())
	{
		int i = opt % c_x_length + 1;
		int j = opt / c_x_length + 1;
		double x, y;
		
		// A
		x = (c_h*(i - 1) + c_h*i) / 2.;
		y = (c_h*(j - 1) + c_h*j) / 2.;
		first1[2 * opt] = first2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		first1[2 * opt + 1] = first2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// B
		x = (c_h*(i + 1) + c_h*i) / 2.;
	    //	y = (c_h*(j - 1) + c_h*j) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second1[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second1[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// C
		//x = (a_x[i + 1] + a_x[i]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		y = (c_h*(j + 1) + c_h*j) / 2.;
		third1[2 * opt] = third2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		third1[2 * opt + 1] = third2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));

		// D 
		x = (c_h*(i - 1) + c_h*i) / 2.;
		//y = (a_y[j + 1] + a_y[j]) / 2.; // это значение совпадает со значением для предыдущей точки значит его можно не расчитывать
		second2[2 * opt] = x - c_tau_b * y * (1. - y) * (c_pi_half + atan(-x));
		second2[2 * opt + 1] = y - c_tau * atan((x - c_lb) * (x - c_rb) * c_tau_to_current_time_level * (y - c_ub) * (y - c_bb));
	}
}

void convert(TriangleResult* result, double* first1, double* second1, double* third1,
	double* first2, double* second2, double* third2, int n)
{
	int k = 0;

	for (int i = 0; i < n; i++)
	{
		result->f[i].first[0] = first1[k];
		result->f[i].first[1] = first1[k + 1];
		result->f[i].second[0] = second1[k];
		result->f[i].second[1] = second1[k + 1];
		result->f[i].third[0] = third1[k];
		result->f[i].third[1] = third1[k + 1];
		result->s[i].first[0] = first2[k];
		result->s[i].first[1] = first2[k + 1];
		result->s[i].second[0] = second2[k];
		result->s[i].second[1] = second2[k + 1];
		result->s[i].third[0] = third2[k];
		result->s[i].third[1] = third2[k + 1];
		k += 2;
	}

}

float get_quad_coord(TriangleResult* result, ComputeParameters* p, int gridSize, int blockSize)
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	size_t size(0), n(0);
	double temp(0);
	n = p->get_inner_matrix_size();


	float elapsedTime;
	double *first1 = NULL, *second1 = NULL, *third1 = NULL, *first2 = NULL, *second2 = NULL, *third2 = NULL;

	// Start record
	hipEventRecord(start, 0);

	hipMemcpyToSymbol(HIP_SYMBOL(c_tau), &p->tau, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_lb), &p->lb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_rb), &p->rb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_bb), &p->bb, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_ub), &p->ub, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(c_n), &n, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(c_x_length), &result->x_length, sizeof(int));
	temp = 1. / (result->x_length + 1);
	hipMemcpyToSymbol(HIP_SYMBOL(c_h), &temp, sizeof(double));

	temp = (1. + p->currentTimeLevel * p->tau) / 10.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_to_current_time_level), &temp, sizeof(double));

	temp = p->b * p->tau;
	hipMemcpyToSymbol(HIP_SYMBOL(c_tau_b), &temp, sizeof(double));

	temp = C_pi_device / 2.;
	hipMemcpyToSymbol(HIP_SYMBOL(c_pi_half), &temp, sizeof(double));
	
	size = 2 * sizeof(double)*n;
	checkCuda(hipMallocManaged(&first1, size));
	checkCuda(hipMallocManaged(&second1, size));
	checkCuda(hipMallocManaged(&third1, size));
	checkCuda(hipMallocManaged(&first2, size));
	checkCuda(hipMallocManaged(&second2, size));
	checkCuda(hipMallocManaged(&third2, size));


	// можно это ядро раскидать на карточки 
	// Вариант 1) На 1 карте считать first1, second1, third1, а на второй считать first2, second2, third2
	// Вариант 2) На 1 карте считать first1, на второй second1 и т. д.
	get_square_coord<< <gridSize, blockSize >> >(first1, second1, third1, first2, second2, third2);
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	convert(result, first1, second1, third1, first2, second2, third2, n);

	hipFree(first1);
	hipFree(second1);
	hipFree(third1);
	hipFree(first2);
	hipFree(second2);
	hipFree(third2);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	hipDeviceReset();
	return elapsedTime;
}